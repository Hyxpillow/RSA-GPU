#include "hip/hip_runtime.h"
#include "obn.h"
#include "hip/hip_runtime.h"

OBN_MUL_GPU_CTX* OBN_MUL_GPU_CTX_new()
{
    OBN_MUL_GPU_CTX *p;
    hipMalloc(&p, sizeof(OBN_MUL_GPU_CTX));
    return p;
}

void OBN_MUL_GPU_CTX_free(OBN_MUL_GPU_CTX* p)
{
    hipFree(p);
}

void OBN_mul_gpu(OURBIGNUM *r, const OURBIGNUM *a, const OURBIGNUM *b, const OBN_MUL_GPU_CTX *ctx)
{
    hipMemcpy(&(ctx->ad), a, sizeof(OURBIGNUM), hipMemcpyHostToDevice);
    hipMemcpy(&(ctx->bd), b, sizeof(OURBIGNUM), hipMemcpyHostToDevice);
    hipMemset(&(ctx->rd), 0, sizeof(OURBIGNUM));
    hipMemset(ctx->buf, 0, sizeof(OURBIGNUM) * sizeof(OURBIGNUM));

    dim3 blockSize(16, 16);
    dim3 gridSize(8, 8);
    f_kernel<<<gridSize, blockSize>>>(ctx->buf, &(ctx->ad), &(ctx->bd));

    for (size_t i = 0; i < OBN_MAX_NUM_BYTES; ++i)
    {
        unsigned int carry = 0;
        for (size_t j = 0; j < OBN_MAX_NUM_BYTES - i; ++j)
        {
            unsigned int product =
                r->data[i + j] +
                buf[i][j]
                carry;

            r->data[i + j] = product & 0xFF;
            carry = product >> 8;
        }
    }

    hipMemcpy(r, &(ctx->buf3), sizeof(OURBIGNUM), hipMemcpyHostToDevice);
}

__global__ void f_kernel(unsigned int **buf, const OURBIGNUM *ad, const OURBIGNUM *bd)
{
    int xxx = 1 + blockIdx.x * blockDim.x + threadIdx.x;
    int yyy = 1 + blockIdx.y * blockDim.y + threadIdx.y;

    for(int i = xxx; i < N - 1; i += gridDim.x * blockDim.x)
    {
        for(int j = yyy; j < N - 1; j += gridDim.y * blockDim.y)
        {
            buf[i][j] = ad->data[i] * bd->data[j];
        }
    }
}

