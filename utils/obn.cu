#include "hip/hip_runtime.h"
#include "obn.h"
#include "hip/hip_runtime.h"

OBN_MUL_GPU_CTX* OBN_MUL_GPU_CTX_new()
{
    OBN_MUL_GPU_CTX *p;
    hipMalloc(&p, sizeof(OBN_MUL_GPU_CTX));
    return p;
}

void OBN_MUL_GPU_CTX_free(OBN_MUL_GPU_CTX* p)
{
    hipFree(p);
}

__global__ void multiply_kernel(int **buf, const OURBIGNUM *ad, const OURBIGNUM *bd)
{
    int xxx = 1 + blockIdx.x * blockDim.x + threadIdx.x;
    int yyy = 1 + blockIdx.y * blockDim.y + threadIdx.y;

    for(int i = xxx; i < OBN_MAX_NUM_BYTES - 1; i += gridDim.x * blockDim.x)
    {
        for(int j = yyy; j < OBN_MAX_NUM_BYTES - 1; j += gridDim.y * blockDim.y)
        {
            buf[i][j] = ad->data[i] * bd->data[j];
        }
    }
}

void OBN_mul_gpu(OURBIGNUM *r, const OURBIGNUM *a, const OURBIGNUM *b, OBN_MUL_GPU_CTX *ctx)
{
    hipMemcpy((void*)(ctx->ad.data), a, sizeof(OURBIGNUM), hipMemcpyHostToDevice);
    hipMemcpy((void*)(ctx->bd.data), b, sizeof(OURBIGNUM), hipMemcpyHostToDevice);
    hipMemset((void*)(ctx->buf), 0, sizeof(OURBIGNUM) * sizeof(OURBIGNUM));

    memset(r->data, 0, sizeof(OURBIGNUM));

    dim3 blockSize(16, 16);
    dim3 gridSize(8, 8);
    multiply_kernel<<<gridSize, blockSize>>>(ctx->buf, &(ctx->ad), &(ctx->bd));

    for (size_t i = 0; i < OBN_MAX_NUM_BYTES; ++i)
    {
        unsigned int carry = 0;
        for (size_t j = 0; j < OBN_MAX_NUM_BYTES - i; ++j)
        {
            unsigned int product =
                r->data[i + j] +
                ctx->buf[i][j] +
                carry;

            r->data[i + j] = product & 0xFF;
            carry = product >> 8;
        }
    }
}

