#include "hip/hip_runtime.h"
#include "obn.h"
#include "hip/hip_runtime.h"

OBN_MUL_GPU_CTX *OBN_MUL_GPU_CTX_new()
{
    OBN_MUL_GPU_CTX *p;
    hipMalloc(&p, sizeof(OBN_MUL_GPU_CTX));
    return p;
}

void OBN_MUL_GPU_CTX_free(OBN_MUL_GPU_CTX *p)
{
    hipFree(p);
}

__global__ void multiply_kernel(OBN_MUL_GPU_CTX* ctx)
{
    int xxx = blockIdx.x * blockDim.x + threadIdx.x;
    int yyy = blockIdx.y * blockDim.y + threadIdx.y;

    for (int i = xxx; i < OBN_MAX_NUM_BYTES; i += gridDim.x * blockDim.x)
    {
        for (int j = yyy; j < OBN_MAX_NUM_BYTES - i; j += gridDim.y * blockDim.y)
        {
            atomicAdd(&(ctx->buf[i + j]), (int)ctx->a[i] * (int)ctx->b[j]);
        }
    }
}

__global__ void split_int32_kernel(OBN_MUL_GPU_CTX* ctx)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = idx; i < OBN_MAX_NUM_BYTES; i += gridDim.x * blockDim.x)
    {
        ctx->low8[i] = ctx->buf[i]  & 0xFF;
        ctx->mid8[i] = (ctx->buf[i] >> 8) & 0xFF;
        ctx->high8[i] = (ctx->buf[i] >> 16) & 0xFF;
    }
}

__global__ void is_carry_all_zero(OBN_MUL_GPU_CTX* ctx)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = idx; i < OBN_MAX_NUM_BYTES; i += gridDim.x * blockDim.x)
        if (ctx->carry[i] != 0)
            ctx->carry_zero_flag = 1; 
}

__global__ void add_kernel(OBN_MUL_GPU_CTX* ctx)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = idx; i < OBN_MAX_NUM_BYTES; i += gridDim.x * blockDim.x)
    {
        int tmp = (int)ctx->x[i] + (int)ctx->y[i];
        ctx->z[i] = tmp & 0xFF;
        if (i + 1 < OBN_MAX_NUM_BYTES)
        {
            ctx->carry[i + 1] = (tmp >> 8) & 0xFF;
        } else {
            ctx->carry[0] = 0;
        }
    }
}

void OBN_mul_gpu(OURBIGNUM *r, const OURBIGNUM *a, const OURBIGNUM *b, OBN_MUL_GPU_CTX *ctx)
{
    hipMemcpy((void *)(ctx->a), a, OBN_MAX_NUM_BYTES, hipMemcpyHostToDevice);
    hipMemcpy((void *)(ctx->b), b, OBN_MAX_NUM_BYTES, hipMemcpyHostToDevice);
    hipMemset((void *)(ctx->buf), 0, sizeof(int) * OBN_MAX_NUM_BYTES);

    int carry_zero_flag;

    dim3 blockSize(16, 16);
    dim3 gridSize(8, 8);
    multiply_kernel<<<gridSize, blockSize>>>(ctx);
    hipDeviceSynchronize();

    // get base, carry8, carry16
    split_int32_kernel<<<8, 32>>>(ctx);
    hipDeviceSynchronize();

    // if carry8 != 0
    hipMemcpy((void *)(ctx->carry), ctx->mid8, OBN_MAX_NUM_BYTES, hipMemcpyDeviceToDevice);
    is_carry_all_zero<<<8, 32>>>(ctx);
    hipDeviceSynchronize();
    hipMemcpy(&carry_zero_flag, (void *)&(ctx->carry_zero_flag), sizeof(int), hipMemcpyDeviceToHost);
    while (carry_zero_flag)
    {
        hipMemcpy((void *)(ctx->x), ctx->low8, OBN_MAX_NUM_BYTES, hipMemcpyDeviceToDevice);
        hipMemcpy((void *)(ctx->y), ctx->carry, OBN_MAX_NUM_BYTES, hipMemcpyDeviceToDevice);

        add_kernel<<<8, 32>>>(ctx);
        hipDeviceSynchronize();

        hipMemcpy((void *)(ctx->low8), ctx->z, OBN_MAX_NUM_BYTES, hipMemcpyDeviceToDevice);

        is_carry_all_zero<<<8, 32>>>(ctx);
        hipDeviceSynchronize();
        hipMemcpy(&carry_zero_flag, (void *)&(ctx->carry_zero_flag), sizeof(int), hipMemcpyDeviceToHost);
    }
    
    // if carry16 != 0
    hipMemcpy((void *)(ctx->carry), ctx->high8, OBN_MAX_NUM_BYTES, hipMemcpyDeviceToDevice);
    is_carry_all_zero<<<8, 32>>>(ctx);
    hipDeviceSynchronize();
    hipMemcpy(&carry_zero_flag, (void *)&(ctx->carry_zero_flag), sizeof(int), hipMemcpyDeviceToHost);
    while (ctx->carry_zero_flag)
    {
        hipMemcpy((void *)(ctx->x), ctx->low8, OBN_MAX_NUM_BYTES, hipMemcpyDeviceToDevice);
        hipMemcpy((void *)(ctx->y), ctx->carry, OBN_MAX_NUM_BYTES, hipMemcpyDeviceToDevice);

        add_kernel<<<8, 32>>>(ctx);
        hipDeviceSynchronize();

        hipMemcpy((void *)(ctx->low8), ctx->z, OBN_MAX_NUM_BYTES, hipMemcpyDeviceToDevice);

        is_carry_all_zero<<<8, 32>>>(ctx);
        hipDeviceSynchronize();
        hipMemcpy(&carry_zero_flag, (void *)&(ctx->carry_zero_flag), sizeof(int), hipMemcpyDeviceToHost);
    }

    hipMemcpy((void *)r, (void *)(ctx->result), OBN_MAX_NUM_BYTES, hipMemcpyDeviceToHost);
}
