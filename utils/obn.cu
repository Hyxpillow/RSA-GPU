#include "hip/hip_runtime.h"
#include "obn.h"
#include "hip/hip_runtime.h"

OBN_MUL_GPU_CTX *OBN_MUL_GPU_CTX_new()
{
    OBN_MUL_GPU_CTX *p;
    hipMalloc(&p, sizeof(OBN_MUL_GPU_CTX));
    return p;
}

void OBN_MUL_GPU_CTX_free(OBN_MUL_GPU_CTX *p)
{
    hipFree(p);
}

__global__ void multiply_kernel(int buf[OBN_MAX_NUM_BYTES][OBN_MAX_NUM_BYTES], const OURBIGNUM *ad, const OURBIGNUM *bd)
{
    int xxx = blockIdx.x * blockDim.x + threadIdx.x;
    int yyy = blockIdx.y * blockDim.y + threadIdx.y;

    for (int i = xxx; i < OBN_MAX_NUM_BYTES; i += gridDim.x * blockDim.x)
    {
        for (int j = yyy; j < OBN_MAX_NUM_BYTES; j += gridDim.y * blockDim.y)
        {
            buf[i][j] = ad->data[i] * bd->data[j];
        }
    }
}

void OBN_mul_gpu(OURBIGNUM *r, const OURBIGNUM *a, const OURBIGNUM *b, OBN_MUL_GPU_CTX *ctx)
{
    int buf[OBN_MAX_NUM_BYTES][OBN_MAX_NUM_BYTES];
    hipMemcpy((void *)(ctx->ad.data), a, sizeof(OURBIGNUM), hipMemcpyHostToDevice);
    hipMemcpy((void *)(ctx->bd.data), b, sizeof(OURBIGNUM), hipMemcpyHostToDevice);
    hipMemset((void *)(ctx->buf), 0, sizeof(int) * OBN_MAX_NUM_BYTES * OBN_MAX_NUM_BYTES);

    dim3 blockSize(16, 16);
    dim3 gridSize(8, 8);
    multiply_kernel<<<gridSize, blockSize>>>(ctx->buf, &(ctx->ad), &(ctx->bd));
    hipDeviceSynchronize();
    hipMemcpy((void *)buf,
               (void *)(ctx->buf),
               sizeof(int) * OBN_MAX_NUM_BYTES * OBN_MAX_NUM_BYTES,
               hipMemcpyDeviceToHost);

    memset(r->data, 0, sizeof(OURBIGNUM));
    for (size_t i = 0; i < OBN_MAX_NUM_BYTES; ++i)
    {
        unsigned int carry = 0;
        for (size_t j = 0; j < OBN_MAX_NUM_BYTES - i; ++j)
        {
            unsigned int product = r->data[i + j] + buf[i][j] + carry;
            r->data[i + j] = product & 0xFF;
            carry = product >> 8;
        }
    }
}
