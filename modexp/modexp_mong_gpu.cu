#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>


/* c: carry (may increment by 1)
   s: partial sum
   x, y: operands */
#define ADD_CARRY(c, s, x, y) \
		do { \
			WORD _t = (x) + (y); \
			(c) += (_t < (x)); \
			(s) = _t; \
		} while (0)

/* Same with ADD_CARRY, but sets y to 0 */
#define ADD_CARRY_CLEAR(c, s, x, y) \
		do { \
			WORD _t = (x) + (y); \
			(y) = 0; \
			(c) += (_t < (x)); \
			(s) = _t; \
		} while (0)

/* b: borrow (may increment by 1)
   d: partial difference
   x, y: operands (a - b) */
#define SUB_BORROW(b, d, x, y) \
		do { \
			WORD _t = (x) - (y); \
			(b) += (_t > (x)); \
			(d) = _t; \
		} while (0)

/* Same with SUB_BORROW, but sets y to 0 */
#define SUB_BORROW_CLEAR(b, d, x, y) \
		do { \
			WORD _t = (x) - (y); \
			(y) = 0; \
			(b) += (_t > (x)); \
			(d) = _t; \
		} while (0)

// Return most significant WORD
__device__ WORD mp_umul_hi(WORD a, WORD b) {
#ifdef USE_64BIT
	return __umul64hi(a, b);
#else
	return __umulhi(a, b);
#endif
}
// Return least significant WORD
__device__ WORD mp_umul_lo(WORD a, WORD b) {
	return a * b;
}


// Montgomery multiplication
__device__ void mp_montmul_dev(WORD *ret, WORD *ar, WORD *br,
	int limb_idx, int idx, int msg_size, WORD *n, WORD np) {
	__shared__ WORD _t[2*BLK_SIZE];
	__shared__ WORD _c[2*BLK_SIZE];

	volatile WORD *t = _t + 2 * msg_size * limb_idx;
	volatile WORD *c = _c + 2 * msg_size * limb_idx;

	c[idx] = 0;
	c[idx + msg_size] = 0;
	t[idx] = 0;
	t[idx + msg_size] = 0;

	for (int i=0; i<msg_size; i++) {
		WORD hi = mp_umul_hi(ar[i], br[idx]);
		WORD lo = mp_umul_lo(ar[i], br[idx]);

		ADD_CARRY(c[i+idx+1], t[i+idx+1], t[i+idx+1], hi);
		ADD_CARRY(c[i+idx], t[i+idx], t[i+idx], lo);

		WORD m = t[i] * np;
		hi = mp_umul_hi(m, n[idx]);
		lo = mp_umul_lo(m, n[idx]);

		ADD_CARRY(c[idx+i+1], t[idx+i+1], t[idx+i+1], hi);
		ADD_CARRY(c[idx+i], t[idx+i], t[idx+i], lo);
		ADD_CARRY_CLEAR(c[idx+i+1], t[idx+i+1], t[idx+i+1], c[idx+i]);
	}

	/* here all t[0] ~ t[msg_size - 1] should be zero. c too */
	while (__any(c[idx + msg_size - 1] != 0))
		ADD_CARRY_CLEAR(c[idx + msg_size], t[idx + msg_size], 
			t[idx + msg_size], c[idx + msg_size - 1]);

	/* step 2: return t or t - n */
	if (c[msg_size * 2 - 1])		// c may be 0 or 1, but not 2
		goto u_is_bigger;

	/* Ugly, but practical.
	 * Can we do this much better with Fermi's ballot()? */
	for (int i = msg_size - 1; i >= 0; i--) {
		if (t[i + msg_size] > n[i])
			goto u_is_bigger;
		if (t[i + msg_size] < n[i])
			goto n_is_bigger;
	}

u_is_bigger:
	/* return t - n. Here, c is used for borrow */
	SUB_BORROW(c[idx], ret[idx], t[idx + msg_size], n[idx]);

	if (idx < msg_size - 1) {
		while (__any(c[idx] != 0)) {
			SUB_BORROW_CLEAR(c[idx + 1], ret[idx + 1],
					ret[idx + 1], c[idx]);
		}
	}
	return;

n_is_bigger:
	/* return t */
	ret[idx] = t[idx + msg_size];
	return;
}

// Exponentiate msg with private key, including Montgomerization and deMontgomerization
__global__ void gpu_modexp(
	int msg_num, int msg_size, WORD *input, WORD *output) {
	__shared__ WORD _ret[BLK_SIZE];
	__shared__ WORD _tmp[BLK_SIZE];

	const int limb_idx = threadIdx.x / msg_size;
	const int msg_idx = blockIdx.x * blockDim.x / msg_size + limb_idx;
	const int idx = threadIdx.x % msg_size;

	if(msg_idx >= msg_num) return;

	WORD *ret = _ret + limb_idx * msg_size;
	WORD *tmp = _tmp + limb_idx * msg_size;

	tmp[idx] = input[msg_idx * msg_size + idx];
	ret[idx] = r_sqr_d[idx];

	mp_montmul_dev(ret, ret, tmp, limb_idx, idx, msg_size, n_d, np_d[0]); /* ret = ar */
	tmp[idx] = ret[idx];

	int t = msg_size * BITS_PER_WORD - 1; /* bit index of d_d */
	while (((d_d[t/BITS_PER_WORD] >> (t%BITS_PER_WORD)) & 1) == 0 && t>0)
		t--;
	t--;

	while (t >= 0) {
		mp_montmul_dev(ret, ret, ret, limb_idx, idx, msg_size, n_d, np_d[0]);
	
		if (((d_d[t/BITS_PER_WORD] >> (t%BITS_PER_WORD)) & 1) == 1) {
			mp_montmul_dev(ret, ret, tmp, limb_idx, idx, msg_size, n_d, np_d[0]);
		}
		t--;
	}

	/* ret = (a^e)*r; calculate a^e = montmul(ret, 1) */
	tmp[idx] = (idx==0);
	mp_montmul_dev(ret, ret, tmp, limb_idx, idx, msg_size, n_d, np_d[0]);

	output[msg_idx*msg_size + idx] = ret[idx];
}